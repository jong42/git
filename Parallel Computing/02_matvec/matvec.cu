#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define DTYPE float

__global__ void kernelAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   //Hier soll die GPU A*x=y berechnen

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse

    for (int j = 0; j < size; j++){
        y[i] += a[i*size+j]*x[j];
	}
		

}



__global__ void kernelAx_sm_versuch1(DTYPE *a, DTYPE *x, DTYPE *y, int size, int k)
{
   //Hier soll die GPU A*x=y mit shared memory berechnen.
	// Funktioniert mit dieser Implementierung nur für Matrizen mit size<=1024 aufgrund maximaler 		// Threadanzahl

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse
    
    __shared__ float sm[1024];
    sm[tid]=0.0;
    __syncthreads();

    // Jeder Thread rechnet ein Produkt aus
    //sm[tid] = a[i*k]*x[tid];
    for (int l=0;l<a[i*k];l++)
    {
		atomicAdd(&sm[tid],x[tid]);
	}
    __syncthreads();
    
    // Für jeden Block werden die Produkte aufsummiert
    for (int j=blockDim.x/2;j>0; j/=2)
    {
		if (tid<j)
		{
			//sm[tid] += sm [tid+j];
			atomicAdd(&sm[tid],sm[tid+j]);
		}
		__syncthreads();
	}		
	if (tid == 0)
	 {
		//y[k] += sm[0];
		atomicAdd(&y[k],sm[0]);
	 }
	 __syncthreads();
}

__global__ void kernelAx_sm_3(DTYPE *a, DTYPE *x, DTYPE *y, int size, int k)
{
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse
    
    __shared__ float sm[512];
    sm[tid]=0.0;
        
	for (int j=0;j<a[tid+k*size];j++)
	{
		atomicAdd(&sm[j],x[tid]);
	}
	__syncthreads();
	
	// Für jeden Block werden die Produkte aufsummiert
    for (int j=blockDim.x/2;j>0; j/=2)
    {
		if (tid<j)
		{
			//sm[tid] += sm [tid+j];
			atomicAdd(&sm[tid],sm[tid+j]);
		}
		__syncthreads();
	}		
	if (tid == 0)
	 {
		//y[k] += sm[0];
		atomicAdd(&y[k],sm[0]);
	 }
	 __syncthreads();
	
	y[k]=sm;
	
}

__global__ void kernelAx_sm_versuch2(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   //Hier soll die GPU A*x=y mit shared memory berechnen.

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse
  
    __shared__ float sm[512];
	sm[tid]=0.0;
	// Jeder Thread rechnet eine Zeile aus
	for (int j = 0; j < size; j++){
        sm[tid] += a[i*size+j]*x[j];
        //atomicAdd(&sm[tid],a[i*size+j]*x[j]);
    }
	__syncthreads();

	// Reduktion
	y[i] = sm[tid];
}





__global__ void kernelATx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   //Hier soll die GPU A^T*x=y berechnen
   

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse

    for (int j = 0; j < size; j++){
        y[i] += a[i+size*j]*x[j];
	}
	
}

__global__ void kernelATx_sm(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   //TODO: Hier soll die GPU A^T*x=y mit shared memory berechnen
   

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse

    for (int j = 0; j < size; j++){
        y[i] += a[i+size*j]*x[j];
	}
	
}


//A mit Werten füllen (hier einfach 1en)
void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++)
      x[i]= (DTYPE)(i+1);
}

void hostAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   // Hier soll der Host A*x=y berechnen
	for (int i = 0;i<size;i++) {
		y[i] = 0;
		for (int j = 0;j<size;j++) {
			y[i] += a[i*size+j] * x[j];
		}
	}
}

void hostATx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   // Hier soll der Host A^T*x=y berechnen
	for (int i = 0;i<size;i++) {
		y[i] = 0;
		for (int j = 0;j<size;j++) {
			y[i] += a[i+size*j] * x[j];
		}
	}
}



bool checkResult(DTYPE *yh_ax, DTYPE *yh_atx, DTYPE *yd_ax, DTYPE *yd_atx, DTYPE *yd_ax_sm, DTYPE *yd_atx_sm, int size)
{
   bool res=true;
   for (int i=0;i<size;i++)
   {
      res&=(yh_ax[i]==yd_ax[i]);
      if (i<10) printf("%f %f %f %f %f %f\n",yh_ax[i],yh_atx[i],yd_ax[i],yd_atx[i],yd_ax_sm		 			[i],yd_atx_sm[i]);
   }
   return res;
}

/*
   Main Routine: 
   Input: i,[threads]
   Berechnet A*x=y auf der GPU wobei A eine Größe von R^{n x n} hat, mit
   n=1024*i
*/
int main(int argc, char**argv)
{
   int i=1;
   int t=512;
   if (argc>1)
   {
      i=atoi(argv[1]);
      if (argc>2) t=atoi(argv[2]);
   }
   else 
   {
      printf("Usage: %s i [threads] \n",argv[0]);
      return -1;
   }
   int size=1024*i;
   //Datenfelder anlegen für Host
   DTYPE *a_host,*x_host, *yh_ax_host, *yh_atx_host, *yd_ax_host, *yd_atx_host, *yd_ax_sm_host, 		*yd_atx_sm_host;
   //und Device
   DTYPE *a_dev, *y_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   //htd: Host->Device Memcpy von A und x
   float htd_time=0.0;
   //dth: Device->Host Memcpy von y
   float dth_time=0.0;
   //kernelA, kernelAT, kernelA_sm
   float kernelA_time=0.0;
   float kernelAT_time=0.0;
   float kernelA_sm_time=0.0;
   float kernelAT_sm_time=0.0;

   //Host Speicher anlegen und A und x füllen
	a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
	x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yh_ax_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yh_atx_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_ax_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_atx_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_ax_sm_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_atx_sm_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	fillA(a_host,size);
	fillX(x_host,size);

   //CUDA Events erstellen
	hipEventCreate(&start);
	hipEventCreate(&end);

   //CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
	hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
	hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
	hipMalloc((void**)&y_dev,size*sizeof(DTYPE));

   //Host->Device Memcpy von A und x + Zeitmessung
	hipEventRecord(start);
    hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
    hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&htd_time,start,end);

   //Konfiguration der CUDA Kernels
	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   dim3 threads(t);
   dim3 grid(size/threads.x);
   
   // kernelAx ausführen und Zeit messen
	hipEventRecord(start);
   	kernelAx<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelA_time,start,end);

   // Device->Host Memcpy für y_dev -> yd_ax_host
	hipEventRecord(start);
  	hipMemcpy(yd_ax_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);
	
	// kernelAx_sm mit shared memory ausführen und Zeit messen
	hipMemset(y_dev, 0,size*sizeof(DTYPE));
	hipEventRecord(start);
	for (int k=0;k<size;k++)
	{
		kernelAx_sm_3<<<grid,threads>>>(a_dev,x_dev,y_dev,size,k);
	}
	//kernelAx_sm<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelA_sm_time,start,end);

   // Device->Host Memcpy für y_dev -> yd_ax_sm_host
	hipEventRecord(start);
  	hipMemcpy(yd_ax_sm_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);

   //: kernelATx ausführen und Zeit messen
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	kernelATx<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelAT_time,start,end); 

   // Device->Host Memcpy für y_dev -> yd_atx_host
	hipEventRecord(start);
  	hipMemcpy(yd_atx_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);

   //: kernelATx_sm mi shared memory ausführen und Zeit messen
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	kernelATx_sm<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelAT_sm_time,start,end); 

   // Device->Host Memcpy für y_dev -> yd_atx_host
	hipEventRecord(start);
  	hipMemcpy(yd_atx_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);

   printf("GPU timing in ms: h->d: %f kernelAx: %f kernelATx: %f kernelAx_sm: %f kernelATx_sm: %f d->h: %f\n",htd_time,kernelA_time,kernelAT_time,kernelA_sm_time,kernelAT_sm_time,dth_time);


   //Nutzen hier timespec um CPU Zeit zu messen
   struct timespec start_h,end_h;
   double hostA_time, hostAT_time;

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start_h);
   //A*x auf Host
	hostAx(a_host, x_host, yh_ax_host, size);

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end_h);
   hostA_time=(double)((end_h.tv_nsec+end_h.tv_sec*1E9) - (start_h.tv_nsec+start_h.tv_sec*1E9))/1E6;
   
   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start_h);
   //A^T*x auf Host
	hostATx(a_host, x_host, yh_atx_host, size);

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end_h);
   hostAT_time=(double)((end_h.tv_nsec+end_h.tv_sec*1E9) - (start_h.tv_nsec+start_h.tv_sec*1E9))/1E6;

   printf("CPU timing in ms: Ax: %f  ATx: %f\n",hostA_time, hostAT_time);

   //checkResult aufrufen
	printf("   CPU_Ax   ;    CPU_Atx   ;    GPU_Ax   ;    GPU_Atx   ;   GPU_Ax_sm  ;  GPU_Atx_sm\n");
	printf("  %f      %f      %f      %f      %f      %f\n",hostA_time, hostAT_time, kernelA_time, 			kernelAT_time, kernelA_sm_time, kernelAT_sm_time);
	checkResult(yh_ax_host, yh_atx_host, yd_ax_host, yd_atx_host, yd_ax_sm_host, yd_atx_sm_host, size);

   //Speicher freigeben (Host UND Device)
   hipFree(a_dev);
   hipFree(x_dev);
   hipFree(y_dev);
   free(a_host);
   free(x_host);
   free(yh_ax_host);
   free(yh_atx_host);
   free(yd_ax_host);
   free(yd_atx_host);
   free(yd_ax_sm_host);
   free(yd_atx_sm_host);

   
   //CUDA Events zerstören
	hipEventDestroy(start);
	hipEventDestroy(end);

   return 0;
}
