#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define DTYPE float

__global__ void kernelAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   //Hier soll die GPU A*x=y berechnen

	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse

    for (int j = 0; j < size; j++){
        y[i] += a[i*size+j]*x[j];
	}
		

}



__global__ void kernelAx_sm(DTYPE *a, DTYPE *x, DTYPE *y, int size, int k)
{
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int gid = tid+bid*bdim;     //Globale Adresse
      
    // shared mem-Grösse gleich der Blockanzahl
    __shared__ float sm[512];
    
    
    // Jeder Thread rechnet ein Zwischenergebnis aus
    sm[tid] = a[gid+k*size]*x[gid];
    __syncthreads();

	// Reduktion
	for (int i=blockDim.x/2;i>0;i=i/2)
	{
       if (tid<i)
           sm[tid]+=sm[tid+i];
       __syncthreads();
	}

	// Übertragen des Ergebnisses
	if (tid==0)
	{
		atomicAdd(&y[k],sm[0]);
	}
	


    

	
}


__global__ void kernelATx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
    //Hier soll die GPU A^T*x=y berechnen
   
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int i = tid+bid*bdim;     //Globale Adresse

    for (int j = 0; j < size; j++){
        y[i] += a[i+size*j]*x[j];
	}
	
}

__global__ void kernelATx_sm(DTYPE *a, DTYPE *x, DTYPE *y, int size, int k)
{
	//Hier soll die GPU A^T*x=y mit shared memory berechnen
	
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int gid = tid+bid*bdim;     //Globale Adresse
      
    // shared mem-Grösse gleich der Blockanzahl
    __shared__ float sm[512];
    
  
    // Jeder Thread rechnet ein Zwischenergebnis aus
    sm[tid] = a[gid*size+k]*x[gid];
    __syncthreads();

	// Reduktion
	for (int i=blockDim.x/2;i>0;i=i/2)
	{
       if (tid<i)
           sm[tid]+=sm[tid+i];
       __syncthreads();
	}

	// Übertragen des Ergebnisses
	if (tid==0)
	{
		atomicAdd(&y[k],sm[0]);
	}
}


__global__ void kernelAx_sm_dp(DTYPE *a, DTYPE *x, DTYPE *y, int size, dim3 grid, dim3 threads)
{	
	//Hier soll die GPU A*x=y mit shared memory und dynamic parallelism berechnen
	
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int gid = tid+bid*bdim;     //Globale Adresse
    
	kernelAx_sm<<<grid,threads>>>(a,x,y,size,gid);
	
}

__global__ void kernelATx_sm_dp(DTYPE *a, DTYPE *x, DTYPE *y, int size, dim3 grid, dim3 threads)
{
	//Hier soll die GPU A^T*x=y mit shared memory und dynamic parallelism berechnen
	
	int tid = threadIdx.x;    //lokaler Thread Index
    int bid = blockIdx.x;     //Index des Blockes
    int bdim= blockDim.x;     //Anzahl an Threads pro Block
    
    int gid = tid+bid*bdim;     //Globale Adresse
    
	kernelATx_sm<<<grid,threads>>>(a,x,y,size,gid);
}

//A mit Werten füllen (hier einfach 1en)
void fillA(DTYPE *a, int size)
{
   for (int i=0;i<size*size;i++)
      a[i]=1.0;
}

//X mit Werten füllen 
void fillX(DTYPE *x, int size)
{
   for (int i=0;i<size;i++)
      x[i]= (DTYPE)(i+1);
}

void hostAx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   // Hier soll der Host A*x=y berechnen
	for (int i = 0;i<size;i++) {
		y[i] = 0;
		for (int j = 0;j<size;j++) {
			y[i] += a[i*size+j] * x[j];
		}
	}
}

void hostATx(DTYPE *a, DTYPE *x, DTYPE *y, int size)
{
   // Hier soll der Host A^T*x=y berechnen
	for (int i = 0;i<size;i++) {
		y[i] = 0;
		for (int j = 0;j<size;j++) {
			y[i] += a[i+size*j] * x[j];
		}
	}
}



bool checkResult(DTYPE *yh_ax, DTYPE *yh_atx, DTYPE *yd_ax, DTYPE *yd_atx, DTYPE *yd_ax_sm, DTYPE *yd_atx_sm, DTYPE *yd_ax_sm_dp, DTYPE *yd_atx_sm_dp, int size)
{
   bool res=true;
   for (int i=0;i<size;i++)
   {
      res&=(yh_ax[i]==yd_ax[i]);
      if (i<10) printf("%f %f %f %f %f %f %f %f\n",yh_ax[i],yh_atx[i],yd_ax[i],yd_atx[i],yd_ax_sm		 			[i],yd_atx_sm[i],yd_ax_sm_dp[i],yd_atx_sm_dp[i]);
   }
   return res;
}

/*
   Main Routine: 
   Input: i,[threads]
   Berechnet A*x=y auf der GPU/CPU wobei A eine Größe von R^{n x n} hat, mit
   n=1024*i
*/
int main(int argc, char**argv)
{
   int i=1;
   int t=512;
   if (argc>1)
   {
      i=atoi(argv[1]);
      if (argc>2) t=atoi(argv[2]);
   }
   else 
   {
      printf("Usage: %s i [threads] \n",argv[0]);
      return -1;
   }
   int size=1024*i;
   //Datenfelder anlegen für Host
   DTYPE *a_host,*x_host, *yh_ax_host, *yh_atx_host, *yd_ax_host, *yd_atx_host, *yd_ax_sm_host, 		*yd_atx_sm_host,*yd_ax_sm_dp_host,*yd_atx_sm_dp_host;
   //und Device
   DTYPE *a_dev, *y_dev,*x_dev;
   //Events für die Zeitmessung
   hipEvent_t start,end;
   //Zeiten: 
   //htd: Host->Device Memcpy von A und x
   float htd_time=0.0;
   //dth: Device->Host Memcpy von y
   float dth_time=0.0;
   //kernelA, kernelAT, kernelA_sm
   float kernelA_time=0.0;
   float kernelAT_time=0.0;
   float kernelA_sm_time=0.0;
   float kernelAT_sm_time=0.0;
   float kernelA_sm_dp_time=0.0;
   float kernelAT_sm_dp_time=0.0;

   //Host Speicher anlegen und A und x füllen
	a_host = (DTYPE*)malloc(size*size*sizeof(DTYPE));
	x_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yh_ax_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yh_atx_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_ax_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_atx_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_ax_sm_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_atx_sm_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_ax_sm_dp_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	yd_atx_sm_dp_host = (DTYPE*)malloc(size*sizeof(DTYPE));
	fillA(a_host,size);
	fillX(x_host,size);

   //CUDA Events erstellen
	hipEventCreate(&start);
	hipEventCreate(&end);

   //CUDA Speicher anlegen für alle Arrays (a_dev,x_dev,y_dev)
	hipMalloc((void**)&a_dev,size*size*sizeof(DTYPE));
	hipMalloc((void**)&x_dev,size*sizeof(DTYPE));
	hipMalloc((void**)&y_dev,size*sizeof(DTYPE));

   //Host->Device Memcpy von A und x + Zeitmessung
	hipEventRecord(start);
    hipMemcpy(a_dev,a_host,size*size*sizeof(DTYPE),hipMemcpyHostToDevice);
    hipMemcpy(x_dev,x_host,size*sizeof(DTYPE),hipMemcpyHostToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&htd_time,start,end);

   //Konfiguration der CUDA Kernels
	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   dim3 threads(t);
   dim3 grid(size/threads.x);
   
   // CacheConfig-Befehle
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAx),hipFuncCachePreferL1);
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelATx),hipFuncCachePreferL1);
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAx_sm),hipFuncCachePreferL1);
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelATx_sm),hipFuncCachePreferL1);
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelAx_sm_dp),hipFuncCachePreferL1);
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelATx_sm_dp),hipFuncCachePreferL1);
   
   // kernelAx ausführen und Zeit messen
	hipEventRecord(start);
   	kernelAx<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelA_time,start,end);

   // Device->Host Memcpy für y_dev -> yd_ax_host
	hipEventRecord(start);
  	hipMemcpy(yd_ax_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);
	
	// kernelAx_sm mit shared memory ausführen und Zeit messen
	hipMemset(y_dev, 0,size*sizeof(DTYPE));
	hipEventRecord(start);
	//printf("xedv: %d", x_dev);
	for (int k=0;k<size;k++)
	{
		kernelAx_sm<<<grid,threads>>>(a_dev,x_dev,y_dev,size,k);
	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelA_sm_time,start,end);

   // Device->Host Memcpy für y_dev -> yd_ax_sm_host
	hipEventRecord(start);
  	hipMemcpy(yd_ax_sm_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);

   //: kernelATx ausführen und Zeit messen
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	kernelATx<<<grid,threads>>>(a_dev,x_dev,y_dev,size);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelAT_time,start,end); 

   // Device->Host Memcpy für y_dev -> yd_atx_host
	hipEventRecord(start);
  	hipMemcpy(yd_atx_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);

   // kernelATx_sm mit shared memory ausführen und Zeit messen
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	for (int k=0;k<size;k++)
	{
		kernelATx_sm<<<grid,threads>>>(a_dev,x_dev,y_dev,size,k);
	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelAT_sm_time,start,end); 

   // Device->Host Memcpy für y_dev -> yd_atx_sm_host
	hipEventRecord(start);
  	hipMemcpy(yd_atx_sm_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);
	
	// kernelAx_sm mit shared memory und dynamic parallelism ausführen und Zeit messen
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	kernelAx_sm_dp<<<grid,threads>>>(a_dev,x_dev,y_dev,size,grid,threads);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelA_sm_dp_time,start,end);
	
	// Device->Host Memcpy für y_dev -> yd_ax_sm_dp_host
	hipEventRecord(start);
  	hipMemcpy(yd_ax_sm_dp_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);
	
	// kernelATx_sm mit shared memory und dynamic parallelism ausführen und Zeit messen
   	
   	hipMemset(y_dev, 0,size*sizeof(DTYPE));
   	hipEventRecord(start);
   	kernelATx_sm_dp<<<grid,threads>>>(a_dev,x_dev,y_dev,size,grid,threads);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&kernelAT_sm_dp_time,start,end);
	
	// Device->Host Memcpy für y_dev -> yd_atx_sm_dp_host
	hipEventRecord(start);
  	hipMemcpy(yd_atx_sm_dp_host,y_dev,size*sizeof(DTYPE),hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&dth_time,start,end);  
	
   printf("GPU timing in ms: h->d: %f kernelAx: %f kernelATx: %f kernelAx_sm: %f kernelATx_sm: %f kernelAx_sm_dp: %f kernelATx_sm_dp: %f d->h: %f\n",htd_time,kernelA_time,kernelAT_time,kernelA_sm_time,kernelAT_sm_time,kernelA_sm_dp_time,kernelAT_sm_dp_time,dth_time);


   //Nutzen hier timespec um CPU Zeit zu messen
   struct timespec start_h,end_h;
   double hostA_time, hostAT_time;

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start_h);
   //A*x auf Host
	hostAx(a_host, x_host, yh_ax_host, size);

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end_h);
   hostA_time=(double)((end_h.tv_nsec+end_h.tv_sec*1E9) - (start_h.tv_nsec+start_h.tv_sec*1E9))/1E6;
   
   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start_h);
   //A^T*x auf Host
	hostATx(a_host, x_host, yh_atx_host, size);

   clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end_h);
   hostAT_time=(double)((end_h.tv_nsec+end_h.tv_sec*1E9) - (start_h.tv_nsec+start_h.tv_sec*1E9))/1E6;

   printf("CPU timing in ms: Ax: %f  ATx: %f\n",hostA_time, hostAT_time);

   //checkResult aufrufen
	printf("   CPU_Ax   ;    CPU_Atx   ;    GPU_Ax   ;    GPU_Atx   ;   GPU_Ax_sm ;  GPU_Atx_sm ;GPU_Ax_sm_dp;GPU_ATx_sm_dp  ;\n");
	printf("  %f      %f      %f      %f      %f      %f    %f     %f\n",hostA_time, hostAT_time, kernelA_time, kernelAT_time, kernelA_sm_time, kernelAT_sm_time,  kernelA_sm_dp_time,  kernelAT_sm_dp_time);
	checkResult(yh_ax_host, yh_atx_host, yd_ax_host, yd_atx_host, yd_ax_sm_host, yd_atx_sm_host, yd_ax_sm_dp_host, yd_atx_sm_dp_host, size);

   //Speicher freigeben (Host UND Device)
   hipFree(a_dev);
   hipFree(x_dev);
   hipFree(y_dev);
   free(a_host);
   free(x_host);
   free(yh_ax_host);
   free(yh_atx_host);
   free(yd_ax_host);
   free(yd_atx_host);
   free(yd_ax_sm_host);
   free(yd_atx_sm_host);

   
   //CUDA Events zerstören
	hipEventDestroy(start);
	hipEventDestroy(end);

   return 0;
}
