#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void kernel(float *a, float *b, int x, int y)
{
   int i=threadIdx.x+blockIdx.x*blockDim.x;
   int j=threadIdx.y+blockIdx.y*blockDim.y;
   __shared__ float sm[256];
   if (i<x && j<y)
   {
      sm[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*x];
      __syncthreads();
      for (int k=blockDim.x/2;k>0;k=k/2)
      {
         if (threadIdx.x<k)
             sm[threadIdx.x+threadIdx.y*blockDim.x]+=sm[threadIdx.x+k+threadIdx.y*blockDim.x];
         __syncthreads();
      }
      if (threadIdx.x==0)
         atomicAdd(&b[j],sm[threadIdx.x+threadIdx.y*blockDim.x]);
   }
}

void initA(float *a, int sx, int sy)
{
    for(int j=0;j<sy;j++)
    {
       for (int i=0;i<sx;i++)
       {
          a[i+j*sx]=(float)(i+1)/(j+1);
       }
    }
}

int checkResults(float *a, float *b, int sx, int sy)
{
   for (int j=0;j<sy;j++)
   {
      float sum=0.0;
      for (int i=0;i<sx;i++)
      {
         sum+=a[i+j*sx];
      }
      if (fabs(b[j] - sum) > 0.0001)
      {
         printf("Error occured in execution in line %i\n",j);
         return(-1);
      }
   }
   return 0;
}

int main()
{
   int sx=64;
   int sy=48;
   float *a,*b,*a_dev,*b_dev;
   //Memory Management
   a=(float*)malloc(sx*sy*sizeof(float));
   b=(float*)malloc(sy*sizeof(float));
   initA(a,sx,sy);
   hipMalloc((void**)&a_dev,sx*sy*sizeof(float));
   hipMalloc((void**)&b_dev,sy*sizeof(float));
   hipMemcpy(a_dev,a,sx*sy*sizeof(float),hipMemcpyHostToDevice);
   hipMemset(b_dev,0,sy*sizeof(float));
   hipError_t err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Memory Management: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }
   //Kernel Execution
   dim3 block(32,8);
   dim3 grid(sx/block.x,sy/block.y);
   kernel<<<grid,block>>>(a_dev,b_dev,sx,sy);
   hipDeviceSynchronize();
   err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Kernel Execution: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }

   hipMemcpy(b,b_dev,sy*sizeof(float),hipMemcpyDeviceToHost);
   int result=checkResults(a,b,sx,sy);
   free(a);
   free(b);
   hipFree(a_dev);
   hipFree(b_dev);
   return result;
}
