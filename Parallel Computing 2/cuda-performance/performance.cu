#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void kernel_atomics(float* a, float *b, int x, int y)
{
   int i=threadIdx.x+blockIdx.x*blockDim.x;
   int j=threadIdx.y+blockIdx.y*blockDim.y;

   if (i<x && j<y)
   {
      atomicAdd(&b[j],a[i+j*x]);
   }
}



__global__ void kernel_shared(float *a, float *b, int x, int y)
{
   int i=threadIdx.x+blockIdx.x*blockDim.x;
   int j=threadIdx.y+blockIdx.y*blockDim.y;
   extern __shared__ float sm[];
   if (i<x && j<y)
   {
      sm[threadIdx.x+threadIdx.y*blockDim.x]=a[i+j*x];
      __syncthreads();
      for (int k=blockDim.x/2;k>0;k/=2)
      {
         if (threadIdx.x<k)
            sm[threadIdx.x+threadIdx.y*blockDim.x]+=sm[threadIdx.x+k+threadIdx.y*blockDim.x];
         __syncthreads();
      }
      if (threadIdx.x==0)
         atomicAdd(&b[j],sm[threadIdx.y*blockDim.x]);
   }
}





#define WARP_SIZE 32
__global__ void kernel_shfl(const float *a, float *b,const int x,const int y)
{
   int i=threadIdx.x+blockIdx.x*blockDim.x;
   int j=threadIdx.y+blockIdx.y*blockDim.y;
   float res=0.0f;
   __shared__ float sm[32];
   if (threadIdx.x<WARP_SIZE)
      sm[threadIdx.x]=0;
   __syncthreads();
   if (j<y)
   {
      for (int k=i;k<x;k+=blockDim.x*gridDim.x)
      {
         res+=a[k+j*x];
      }
      #pragma unroll
      for (int mask=WARP_SIZE/2;mask>0;mask>>=1)
      {
         res+=__shfl_down(res,mask);
      }
      if (threadIdx.x%WARP_SIZE==0)
      {
         sm[threadIdx.x/WARP_SIZE]=res;
      }
      __syncthreads();
      if (threadIdx.x<WARP_SIZE)
      {
         res=sm[threadIdx.x];
         #pragma unroll
         for (int mask=WARP_SIZE/2;mask>0;mask>>=1)
         {
            res+=__shfl_down(res,mask);
         }
      }
      if (threadIdx.x==0)
      {
         //atomicAdd(&b[j],res);
         b[j]=res;
      }
   }
}












void initA(float *a, int sx, int sy)
{
    for(int j=0;j<sy;j++)
    {
       for (int i=0;i<sx;i++)
       {
          a[i+j*sx]=(float)(i+1)/(j+1);
       }
    }
}

int checkResults(float *a, float *b, int sx, int sy)
{
   float error=0.0;
   for (int j=0;j<sy;j++)
   {
      float sum=0.0;
      for (int i=0;i<sx;i++)
      {
         sum+=a[i+j*sx];
      }
      error=max(error,abs(1-(b[j]/sum)));
      if (abs(1-(b[j]/sum))>1e-4)
      {
         printf("Error occured in execution in line %i (%f!=%f)\n",j,b[j],sum);
         return(-1);
      }
   }
   printf("Test passed! Max relative error was %f\n",error);
   return 0;
}

int main(int argc, char ** argv)
{
   int sx=8192;
   int sy=4096;
   if (argc>2)
   {
      sx=(int)atoi(argv[1]);
      sy=(int)atoi(argv[2]);
   }
   float *a,*b,*a_dev,*b_dev;
   //Memory Management
   a=(float*)malloc(sx*sy*sizeof(float));
   b=(float*)malloc(sy*sizeof(float));
   initA(a,sx,sy);
   hipMalloc((void**)&a_dev,sx*sy*sizeof(float));
   hipMalloc((void**)&b_dev,sy*sizeof(float));
   hipMemcpy(a_dev,a,sx*sy*sizeof(float),hipMemcpyHostToDevice);
   hipMemset(b_dev,0,sy*sizeof(float));
   hipError_t err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Memory Management: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }
   dim3 block(1,256);
   dim3 grid((int)ceilf((float)sx/block.x),(int)ceilf((float)sy/block.y));
   //Kernel Execution Atomics Global
   hipMemset(b_dev,0,sy*sizeof(float));
   kernel_atomics<<<grid,block>>>(a_dev,b_dev,sx,sy);
   hipDeviceSynchronize();
   err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Atomics Kernel Execution: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }

   hipMemcpy(b,b_dev,sy*sizeof(float),hipMemcpyDeviceToHost);
   int result=checkResults(a,b,sx,sy);

   //Kernel Execution SM
   block.x=32;
   block.y=8;
   grid.x=(int)ceilf((float)sx/block.x);
   grid.y=(int)ceilf((float)sy/block.y);
   hipMemset(b_dev,0,sy*sizeof(float));
   kernel_shared<<<grid,block,block.x*block.y*sizeof(float)>>>(a_dev,b_dev,sx,sy);
   hipDeviceSynchronize();
   err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Shared Kernel Execution: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }

   hipMemcpy(b,b_dev,sy*sizeof(float),hipMemcpyDeviceToHost);
   result=checkResults(a,b,sx,sy);

   //execute performance optimized code

   //Use shuffle instead of sm and 1 Block in x direction 
   hipMemset(b_dev,0,sy*sizeof(float));
   block.x=128;
   block.y=1;
   grid.x=1;
   grid.y=sy;
   hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_shfl),hipFuncCachePreferL1);
   kernel_shfl<<<grid,block>>>(a_dev,b_dev,sx,sy);
   hipDeviceSynchronize();
   err=hipGetLastError();
   if (err!=hipSuccess)
   {
      printf("An Error occured in Shuffle Kernel Execution: %s (%i)\n",hipGetErrorString(err),err);
      return(-1);
   }

   hipMemcpy(b,b_dev,sy*sizeof(float),hipMemcpyDeviceToHost);
   result=checkResults(a,b,sx,sy);
   free(a);
   free(b);
   hipFree(a_dev);
   hipFree(b_dev);
   return result;
}
