#include <stdio.h>
#include <hip/hip_runtime.h>

//device code
//Gemeinsamer Speicher der Threads
//__shared__ float c[1024];

__global__ void mykernel(float *a,float *b,float *c) {
// perform calculation
int index = threadIdx.x + blockIdx.x *blockDim.x;
c[index] = a[index]*b[index];
//printf("%f\n",c[index]);
// Barriere zur Synchronisation von Threads
__syncthreads();
}

//host
int main (){
	
	//Allocate vectors in host memory
	int threadnr = 1024;
	float n = 524288;
	float *h_a, *h_b, *h_c;
	h_a = (float *)malloc(sizeof(float) * n);
	h_b = (float *)malloc(sizeof(float) * n);
	h_c = (float *)malloc(sizeof(float) * n);
	
	//Create vector values
	int i;
	for (i=0;i<n;i++)
		{
			h_a[i] = i+1;
			h_b[i] = n-i;
		}
	
	//Allocate vectors in device memory
	float *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, n*sizeof(float));
	hipMalloc((void **)&d_b, n*sizeof(float));
	hipMalloc((void **)&d_c, n*sizeof(float));
	//Copy vectors from host memory to device memory
	hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n*sizeof(float), hipMemcpyHostToDevice);

	//Invoke kernel
	mykernel<<<n/threadnr, threadnr>>>(d_a, d_b, d_c);
	//Copy result from device memory to host memory
	hipMemcpy(h_c, d_c, n*sizeof(float), hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	// calculate sum of al c elements
	float sum = 0;	
	for (i=0;i<n;i++)
		{ 
			sum = sum + h_c[i];
		}

	printf("sum: %f\t", sum);
	
	return 0;
}

