#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc -arch=sm_61 vdp.cu -o vdp
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>
#define N (524288)
//#define N (2097152)
//#define N (4194304)
#define THREADS_PER_BLOCK (128)
#define NUM_BLOCKS (N/THREADS_PER_BLOCK)

__global__ void dot_product(float *d_A, float *d_B, float *d_C){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float se[THREADS_PER_BLOCK];
    se[threadIdx.x] = (float)d_A[index] * d_B[index];
    __syncthreads();
    if (threadIdx.x == 0){
        float sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK;i++){

            sum += se[i];
        }
        // No time to waste
        atomicAdd(d_C,  sum);
    }
}

int main(){

    clock_t start, end;
    float cpu_time_used;

    float *h_A, *h_B, *d_A, *d_B;
    float *h_C, *d_C;
    int size = N * sizeof(float);
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, sizeof(float));
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(sizeof(float));

    // value creation for dot product
    for (int i = 0; i < N; i++){
        h_A[i] = i+1;
        h_B[i] = N-i;
    }
    *h_C = 0;

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(float), hipMemcpyHostToDevice);
    start = clock();
    dot_product<<< NUM_BLOCKS, THREADS_PER_BLOCK >> >(d_A, d_B, d_C);
    end = clock();
    hipMemcpy(h_C, d_C, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceReset();
    cpu_time_used = ((float) (end - start)) / CLOCKS_PER_SEC;
    printf("CUDA:\t\t%f in %f\n", *h_C, cpu_time_used);

    // Never trust CUDA
    start = clock();
    float sumCheck = 0;
    for (int i = 0; i < N; i++){

        sumCheck += (float)h_A[i] * h_B[i];
    }
    end = clock();
    cpu_time_used = ((float) (end - start)) / CLOCKS_PER_SEC;
    printf("CPU Check:\t%f in %f\n", sumCheck, cpu_time_used);
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
 }
