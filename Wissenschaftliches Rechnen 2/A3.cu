
#include <hip/hip_runtime.h>
__device__ volatile int X = 1, Y = 2;   // volatile Variable kann außerhalb des Programms 
										// verändert werden
__device__ void schreibe() {
	X = 10;
	Y = 20;
}
__device__ void lese() {
	int A = X;
	int B = Y;
}


// A kann die Werte 1 oder 10 annehmen, B die Werte 2 oder 20
